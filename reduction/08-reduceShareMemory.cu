
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


/*
GPU程序执行时间: 11.775000 ms
*/

#define DIM  512

// CUDA核函数，执行并行规约操作
__global__ void reduceSmemUnroll(int* g_odata, int* g_idata, unsigned int n) {

    __shared__ int smem[DIM];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    
    int tmpSum = 0;
    //一个block处理8个数据块，将其余7个数据块的数据累加到第一个数据块上
    if(idx + 7*blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2*blockDim.x];
        int a4 = g_idata[idx + 3*blockDim.x];
        int b1 = g_idata[idx + 4*blockDim.x];
        int b2 = g_idata[idx + 5*blockDim.x];
        int b3 = g_idata[idx + 6*blockDim.x];
        int b4 = g_idata[idx + 7*blockDim.x];
    
        tmpSum = a1+a2+a3+a4+b1+b2+b3+b4;
    }

    smem[tid] = tmpSum;
    __syncthreads(); //必须加上，不然结果不对。

    if(blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if(blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if(blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if(blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    //unrolling warp
    if(tid < 32) {
        volatile int *vmem = smem;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
    
    if(tid == 0) g_odata[blockIdx.x] = smem[0];
}

int recursiveReduce(int *data, int const size) {
    if(size == 1) return data[0];
    int const stride = size / 2;
    for(int i=0; i<stride; i++) {
        data[i] += data[i + stride];
    }    

    return recursiveReduce(data, stride);
}


int main(int argc, char **argv) {
    
    struct timeval time_start,  time_stop ;
    double time_pass;

    int size = 1<<24;
    int blockSize = 512;
    if(argc > 1) {
        blockSize = atoi(argv[1]);
    }

    dim3 block (blockSize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x/8*sizeof(int));
    int *tmp = (int *)malloc(bytes);

    for(int i=0; i<size; i++) {
        h_idata[i] = (int) (rand() & 0xFF);
    }

    memcpy(tmp, h_idata, bytes);

    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void**)&d_idata, bytes);
    hipMalloc((void**)&d_odata, grid.x*sizeof(int));

    //cpu reduction
    gettimeofday(&time_start, NULL);
    int cpu_sum = recursiveReduce(tmp, size);
    gettimeofday(&time_stop, NULL);

    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("CPU程序执行时间: %lf ms\n", time_pass);

    gettimeofday(&time_start, NULL);

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceSmemUnroll<<<grid.x/8, block.x>>>(d_odata, d_idata, size);
    hipMemcpy(h_odata, d_odata, grid.x/8 * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    gettimeofday(&time_stop, NULL);
    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("GPU程序执行时间: %lf ms\n", time_pass);

    int gpu_sum = 0;
    for(int i=0; i<grid.x/8; i++) {
        gpu_sum += h_odata[i];
    }

    if(cpu_sum == gpu_sum) {
        printf("CPU&GPU have same result!!! \n");
    } else {
        printf("CPU&GPU result are diffrent!!! \n");
        printf("cpu_sum=%d \n",cpu_sum);
        printf("gpu_sum=%d \n",gpu_sum);

    }

    free(h_idata);
    free(h_odata);

    hipFree(d_idata);
    hipFree(d_odata);


    return 0;
}
