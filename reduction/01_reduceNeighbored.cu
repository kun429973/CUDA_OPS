
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


/*
GPU编程逻辑：
    第一次循环中每个block中相邻两个thread上的数相加
    第二次循环中相隔1个的两个thread上的数相加
    依次循环
    最后所有block中的数和都相加到threadIdx=0上面了。
要求:
    每个block中的线程数为偶数。
*/

// CUDA核函数，执行并行规约操作
__global__ void reduceNeighbored(int* g_odata, int* g_idata, unsigned int n) {

    int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    if(tid >= n) return;
    for(int stride=1; stride < blockDim.x; stride *= 2) {
        if((tid %(2 * stride)) == 0) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0) g_odata[blockIdx.x] = idata[0];
    
}

int recursiveReduce(int *data, int const size) {
    if(size == 1) return data[0];
    int const stride = size / 2;
    for(int i=0; i<stride; i++) {
        data[i] += data[i + stride];
    }    

    return recursiveReduce(data, stride);
}


int main(int argc, char **argv) {
    
    struct timeval time_start,  time_stop ;
    double time_pass;

    int size = 1<<24;
    int blockSize = 512;
    if(argc > 1) {
        blockSize = atoi(argv[1]);
    }

    dim3 block (blockSize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x*sizeof(int));
    int *tmp = (int *)malloc(bytes);

    for(int i=0; i<size; i++) {
        h_idata[i] = (int) (rand() & 0xFF);
    }

    memcpy(tmp, h_idata, bytes);

    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void**)&d_idata, bytes);
    hipMalloc((void**)&d_odata, grid.x*sizeof(int));

    //cpu reduction
    gettimeofday(&time_start, NULL);
    int cpu_sum = recursiveReduce(tmp, size);
    gettimeofday(&time_stop, NULL);

    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("CPU程序执行时间: %lf ms\n", time_pass);

    gettimeofday(&time_start, NULL);

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceNeighbored<<<grid, block>>>(d_odata, d_idata, size);
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday(&time_stop, NULL);
    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("GPU程序执行时间: %lf ms\n", time_pass);

    int gpu_sum = 0;
    for(int i=0; i<grid.x; i++) {
        gpu_sum += h_odata[i];
    }

    if(cpu_sum == gpu_sum) {
        printf("CPU&GPU have same result!!! \n");
    } else {
        printf("CPU&GPU result are diffrent!!! \n");
    }

    free(h_idata);
    free(h_odata);

    hipFree(d_idata);
    hipFree(d_odata);


    return 0;
}
