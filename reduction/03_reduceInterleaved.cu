
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


/*
GPU程序思路：每个block中的threads/2,后半部分加到前半部分上。
GPU程序执行时间: 12.734000 ms  
*/

// CUDA核函数，执行并行规约操作
__global__ void reduceInterleaved(int* g_odata, int* g_idata, unsigned int n) {

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int *idata = g_idata + blockIdx.x * blockDim.x;

    if(idx >= n) return;
    for(int stride=blockDim.x / 2; stride > 0; stride >>= 1) {
        if(tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0) g_odata[blockIdx.x] = idata[0];
    
}

int recursiveReduce(int *data, int const size) {
    if(size == 1) return data[0];
    int const stride = size / 2;
    for(int i=0; i<stride; i++) {
        data[i] += data[i + stride];
    }    

    return recursiveReduce(data, stride);
}


int main(int argc, char **argv) {
    
    struct timeval time_start,  time_stop ;
    double time_pass;

    int size = 1<<24;
    int blockSize = 512;
    if(argc > 1) {
        blockSize = atoi(argv[1]);
    }

    dim3 block (blockSize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x*sizeof(int));
    int *tmp = (int *)malloc(bytes);

    for(int i=0; i<size; i++) {
        h_idata[i] = (int) (rand() & 0xFF);
    }

    memcpy(tmp, h_idata, bytes);

    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void**)&d_idata, bytes);
    hipMalloc((void**)&d_odata, grid.x*sizeof(int));

    //cpu reduction
    gettimeofday(&time_start, NULL);
    int cpu_sum = recursiveReduce(tmp, size);
    gettimeofday(&time_stop, NULL);

    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("CPU程序执行时间: %lf ms\n", time_pass);

    gettimeofday(&time_start, NULL);

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceInterleaved<<<grid, block>>>(d_odata, d_idata, size);
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday(&time_stop, NULL);
    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("GPU程序执行时间: %lf ms\n", time_pass);

    int gpu_sum = 0;
    for(int i=0; i<grid.x; i++) {
        gpu_sum += h_odata[i];
    }

    if(cpu_sum == gpu_sum) {
        printf("CPU&GPU have same result!!! \n");
    } else {
        printf("CPU&GPU result are diffrent!!! \n");
    }

    free(h_idata);
    free(h_odata);

    hipFree(d_idata);
    hipFree(d_odata);


    return 0;
}
