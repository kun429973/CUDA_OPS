
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


/*
相比于  06-reduceCompletrUnrollWarps8.cu，这里使用了模板参数替换了块的大小，检查块大小的if语句在编译时被评估，
如果这一条为false，那么编译时它将会被删除，使得内循环更有效率。
GPU程序执行时间: 11.558000 ms
*/

// CUDA核函数，执行并行规约操作
template<unsigned int iBlockSize>
__global__ void reduceCompleteUnroll(int* g_odata, int* g_idata,  int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    //一个block处理8个数据块，将其余7个数据块的数据累加到第一个数据块上
    if(idx + 7*blockDim.x < n) {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2*blockDim.x];
        int a4 = g_idata[idx + 3*blockDim.x];
        int b1 = g_idata[idx + 4*blockDim.x];
        int b2 = g_idata[idx + 5*blockDim.x];
        int b3 = g_idata[idx + 6*blockDim.x];
        int b4 = g_idata[idx + 7*blockDim.x];
    
        g_idata[idx] = a1+a2+a3+a4+b1+b2+b3+b4;
    }
    __syncthreads(); //必须加上，不然结果不对。

    if(iBlockSize >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if(iBlockSize >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if(iBlockSize >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if(iBlockSize >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();
    
    //unrolling warp
    if(tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
    
    if(tid == 0) g_odata[blockIdx.x] = idata[0];
}

int recursiveReduce(int *data, int const size) {
    if(size == 1) return data[0];
    int const stride = size / 2;
    for(int i=0; i<stride; i++) {
        data[i] += data[i + stride];
    }    

    return recursiveReduce(data, stride);
}


int main(int argc, char **argv) {
    
    struct timeval time_start,  time_stop ;
    double time_pass;

    int size = 1<<24;
    int blockSize = 512;
    if(argc > 1) {
        blockSize = atoi(argv[1]);
    }

    dim3 block (blockSize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x/8*sizeof(int));
    int *tmp = (int *)malloc(bytes);

    for(int i=0; i<size; i++) {
        h_idata[i] = (int) (rand() & 0xFF);
    }

    memcpy(tmp, h_idata, bytes);

    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void**)&d_idata, bytes);
    hipMalloc((void**)&d_odata, grid.x*sizeof(int));

    //cpu reduction
    gettimeofday(&time_start, NULL);
    int cpu_sum = recursiveReduce(tmp, size);
    gettimeofday(&time_stop, NULL);

    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("CPU程序执行时间: %lf ms\n", time_pass);

    gettimeofday(&time_start, NULL);

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceCompleteUnroll<512><<<grid.x/8, block.x>>>(d_odata, d_idata, size);
    hipMemcpy(h_odata, d_odata, grid.x/8 * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    gettimeofday(&time_stop, NULL);
    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("GPU程序执行时间: %lf ms\n", time_pass);

    int gpu_sum = 0;
    for(int i=0; i<grid.x/8; i++) {
        gpu_sum += h_odata[i];
    }

    if(cpu_sum == gpu_sum) {
        printf("CPU&GPU have same result!!! \n");
    } else {
        printf("CPU&GPU result are diffrent!!! \n");
        printf("cpu_sum=%d \n",cpu_sum);
        printf("gpu_sum=%d \n",gpu_sum);

    }

    free(h_idata);
    free(h_odata);

    hipFree(d_idata);
    hipFree(d_odata);


    return 0;
}
