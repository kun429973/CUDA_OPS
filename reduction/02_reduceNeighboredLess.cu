
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>


/*
二，针对01_reduceNeighbored.cu 中 if((tid %(2 * stride)) == 0) 线程束分化严重，
优化：使用 int index = 2 * stride * tid;防止线程束的分化，线程0->data[0],线程1->data[2] ...
512个线程，16个线程束，仅使用前8个线程束就可以了。

GPU程序执行时间: 13.580000 ms    
*/

// CUDA核函数，执行并行规约操作
__global__ void reduceNeighboredLess(int* g_odata, int* g_idata, unsigned int n) {

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int *idata = g_idata + blockIdx.x * blockDim.x;

    if(idx >= n) return;
    for(int stride=1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if(index < blockDim.x) {
            idata[index] += idata[index + stride];
        }
        __syncthreads();
    }

    if(tid == 0) g_odata[blockIdx.x] = idata[0];
    
}

int recursiveReduce(int *data, int const size) {
    if(size == 1) return data[0];
    int const stride = size / 2;
    for(int i=0; i<stride; i++) {
        data[i] += data[i + stride];
    }    

    return recursiveReduce(data, stride);
}


int main(int argc, char **argv) {
    
    struct timeval time_start,  time_stop ;
    double time_pass;

    int size = 1<<24;
    int blockSize = 512;
    if(argc > 1) {
        blockSize = atoi(argv[1]);
    }

    dim3 block (blockSize, 1);
    dim3 grid ((size+block.x-1)/block.x, 1);

    size_t bytes = size * sizeof(int);
    int *h_idata = (int *)malloc(bytes);
    int *h_odata = (int *)malloc(grid.x*sizeof(int));
    int *tmp = (int *)malloc(bytes);

    for(int i=0; i<size; i++) {
        h_idata[i] = (int) (rand() & 0xFF);
    }

    memcpy(tmp, h_idata, bytes);

    int *d_idata = NULL;
    int *d_odata = NULL;
    hipMalloc((void**)&d_idata, bytes);
    hipMalloc((void**)&d_odata, grid.x*sizeof(int));

    //cpu reduction
    gettimeofday(&time_start, NULL);
    int cpu_sum = recursiveReduce(tmp, size);
    gettimeofday(&time_stop, NULL);

    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("CPU程序执行时间: %lf ms\n", time_pass);

    gettimeofday(&time_start, NULL);

    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    reduceNeighboredLess<<<grid, block>>>(d_odata, d_idata, size);
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);

    gettimeofday(&time_stop, NULL);
    time_pass = ((double)time_stop.tv_sec -(double)time_start.tv_sec)*1000 + ((double)time_stop.tv_usec-(double)time_start.tv_usec)/1000;
    printf("GPU程序执行时间: %lf ms\n", time_pass);

    int gpu_sum = 0;
    for(int i=0; i<grid.x; i++) {
        gpu_sum += h_odata[i];
    }

    if(cpu_sum == gpu_sum) {
        printf("CPU&GPU have same result!!! \n");
    } else {
        printf("CPU&GPU result are diffrent!!! \n");
    }

    free(h_idata);
    free(h_odata);

    hipFree(d_idata);
    hipFree(d_odata);


    return 0;
}
